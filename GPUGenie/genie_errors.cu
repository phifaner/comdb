#include "hip/hip_runtime.h"
/*! \file genie_errors.cu
 *  \brief Implementation for genie_errors.h
 */

#include "Logger.h"

#include "genie_errors.h"

GPUGenie::genie_error::genie_error(const char * msg): std::runtime_error(msg){
	Logger::log(Logger::ALERT, "%s", msg);
}





